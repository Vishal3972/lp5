#include "hip/hip_runtime.h"
// Program1

#include <hip/hip_runtime.h>
#include <iostream>

// CUDA kernel to perform matrix multiplication
__global__ void matmul(int *A, int *B, int *C, int N)
{
    int Row = blockIdx.y * blockDim.y + threadIdx.y; // Calculate the row index
    int Col = blockIdx.x * blockDim.x + threadIdx.x; // Calculate the column index

    // Ensure the thread is within the matrix dimensions
    if (Row < N && Col < N)
    {
        int Pvalue = 0; // Initialize the element of the resulting matrix

        // Perform dot product of row of A and column of B to compute element of C
        for (int k = 0; k < N; k++)
        {
            Pvalue += A[Row * N + k] * B[k * N + Col];
        }

        C[Row * N + Col] = Pvalue; // Store the computed element in the result matrix
    }
}

int main()
{
    int N = 512;                    // Size of the matrices
    int size = N * N * sizeof(int); // Size of each matrix in bytes
    int *A, *B, *C;                 // Host matrices
    int *dev_A, *dev_B, *dev_C;     // Device matrices
    hipHostMalloc(&A, size);       // Allocate pinned memory for matrix A
    hipHostMalloc(&B, size);       // Allocate pinned memory for matrix B
    hipHostMalloc(&C, size);       // Allocate pinned memory for matrix C
    hipMalloc(&dev_A, size);       // Allocate memory for matrix A on the device
    hipMalloc(&dev_B, size);       // Allocate memory for matrix B on the device
    hipMalloc(&dev_C, size);       // Allocate memory for matrix C on the device

    // Initialize matrices A and B
    for (int i = 0; i < N; i++)
    {
        for (int j = 0; j < N; j++)
        {
            A[i * N + j] = i * N + j; // Initialize matrix A with row-major order
            B[i * N + j] = j * N + i; // Initialize matrix B with column-major order
        }
    }

    // Copy matrices A and B from host to device
    hipMemcpy(dev_A, A, size, hipMemcpyHostToDevice);
    hipMemcpy(dev_B, B, size, hipMemcpyHostToDevice);

    // Define thread block dimensions and grid dimensions
    dim3 dimBlock(16, 16);                        // 16x16 thread block
    dim3 dimGrid(N / dimBlock.x, N / dimBlock.y); // N/16 x N/16 grid

    // Launch the kernel for matrix multiplication
    matmul<<<dimGrid, dimBlock>>>(dev_A, dev_B, dev_C, N);

    // Copy the result matrix C from device to host
    hipMemcpy(C, dev_C, size, hipMemcpyDeviceToHost);

    // Print the first 10x10 elements of the result matrix C
    for (int i = 0; i < 10; i++)
    {
        for (int j = 0; j < 10; j++)
        {
            std::cout << C[i * N + j] << " ";
        }
        std::cout << std::endl;
    }

    // Free memory on the device
    hipFree(dev_A);
    hipFree(dev_B);
    hipFree(dev_C);
    // Free pinned memory on the host
    hipHostFree(A);
    hipHostFree(B);
    hipHostFree(C);

    return 0;
}

// Program 2

#include <iostream>
#include <hip/hip_runtime.h>

using namespace std;

// CUDA kernel to add two vectors
__global__ void addVectors(int *A, int *B, int *C, int n)
{
    // Calculate the global index of the thread
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n)
    {
        // Add corresponding elements from A and B and store the result in C
        C[i] = A[i] + B[i];
    }
}

int main()
{
    int n = 1000000;            // Size of the vectors
    int *A, *B, *C;             // Host vectors
    int size = n * sizeof(int); // Size of each vector in bytes

    // Allocate pinned memory on the host for vectors A, B, and C
    hipHostMalloc(&A, size);
    hipHostMalloc(&B, size);
    hipHostMalloc(&C, size);

    // Initialize vectors A and B
    for (int i = 0; i < n; i++)
    {
        A[i] = i;
        B[i] = i * 2;
    }

    // Allocate memory for vectors A, B, and C on the device
    int *dev_A, *dev_B, *dev_C;
    hipMalloc(&dev_A, size);
    hipMalloc(&dev_B, size);
    hipMalloc(&dev_C, size);

    // Copy vectors A and B from host to device
    hipMemcpy(dev_A, A, size, hipMemcpyHostToDevice);
    hipMemcpy(dev_B, B, size, hipMemcpyHostToDevice);

    // Launch the kernel to add vectors A and B
    int blockSize = 256;
    int numBlocks = (n + blockSize - 1) / blockSize;
    addVectors<<<numBlocks, blockSize>>>(dev_A, dev_B, dev_C, n);

    // Copy vector C from device to host
    hipMemcpy(C, dev_C, size, hipMemcpyDeviceToHost);

    // Print the first 10 elements of the result vector C
    for (int i = 0; i < 10; i++)
    {
        cout << C[i] << " ";
    }
    cout << endl;

    // Free memory on the device
    hipFree(dev_A);
    hipFree(dev_B);
    hipFree(dev_C);
    // Free pinned memory on the host
    hipHostFree(A);
    hipHostFree(B);
    hipHostFree(C);

    return 0;
}
